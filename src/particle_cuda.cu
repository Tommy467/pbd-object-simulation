#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <cassert>
// #include <__msvc_ostream.hpp>
#include <iostream>
#include <fstream>

#include "utils.hpp"
#include "object.hpp"

#ifdef USE_GPU

#ifdef OUTPUT_RESULTS
extern std::ofstream output_file;
#endif
extern int32_t particle_min_count;

constexpr int num_cell = 8;
struct Grids_Cuda {
    int32_t *grid_index;
    int32_t *object_index;
    int32_t *object_counts;
    int32_t grid_count;
} grids;

void Object_initDeviceMemory(Object *objects) {
    if (objects->device_allocated) return;
    hipMalloc(&objects->d_position_x,      sizeof(float) * N);
    hipMalloc(&objects->d_position_y,      sizeof(float) * N);
    hipMalloc(&objects->d_last_position_x, sizeof(float) * N);
    hipMalloc(&objects->d_last_position_y, sizeof(float) * N);
    hipMalloc(&objects->d_radius,          sizeof(float) * N);
    objects->device_allocated = true;
}

void Object_freeDeviceMemory(Object *objects) {
    if (!objects->device_allocated) return;
    hipFree(objects->d_position_x);
    hipFree(objects->d_position_y);
    hipFree(objects->d_last_position_x);
    hipFree(objects->d_last_position_y);
    hipFree(objects->d_radius);
    objects->device_allocated = false;
}

void Grids_initDeviceMemory(const int32_t world_width, const int32_t world_height) {
    const int32_t size = world_width * world_height;
    hipMalloc(&grids.grid_index, sizeof(int32_t) * size);
    hipMalloc(&grids.object_index, sizeof(int32_t) * size * num_cell);
    hipMalloc(&grids.object_counts, sizeof(int32_t) * size);
    grids.grid_count = size;
}

void Grids_freeDeviceMemory() {
    hipFree(grids.grid_index);
    hipFree(grids.object_index);
    hipFree(grids.object_counts);
}

void objectCopyToDevice(const Object *objects) {
    if (!objects->device_allocated) return;
    hipMemcpy(objects->d_position_x,      objects->position_x,      sizeof(float) * objects->size, hipMemcpyHostToDevice);
    hipMemcpy(objects->d_position_y,      objects->position_y,      sizeof(float) * objects->size, hipMemcpyHostToDevice);
    hipMemcpy(objects->d_last_position_x, objects->last_position_x, sizeof(float) * objects->size, hipMemcpyHostToDevice);
    hipMemcpy(objects->d_last_position_y, objects->last_position_y, sizeof(float) * objects->size, hipMemcpyHostToDevice);
    hipMemcpy(objects->d_radius,          objects->radius,          sizeof(float) * objects->size, hipMemcpyHostToDevice);
}

void objectCopyToHost(Object *objects) {
    if (!objects->device_allocated) return;
    hipMemcpy(objects->position_x,      objects->d_position_x,      sizeof(float) * objects->size, hipMemcpyDeviceToHost);
    hipMemcpy(objects->position_y,      objects->d_position_y,      sizeof(float) * objects->size, hipMemcpyDeviceToHost);
    hipMemcpy(objects->last_position_x, objects->d_last_position_x, sizeof(float) * objects->size, hipMemcpyDeviceToHost);
    hipMemcpy(objects->last_position_y, objects->d_last_position_y, sizeof(float) * objects->size, hipMemcpyDeviceToHost);
}

__global__ void updateObjects_kernel(
    float *position_x, float *position_y,
    float *last_position_x, float *last_position_y,
    const float acceleration_x, const float acceleration_y,
    const float *radius,
    const int size, const float delta_time, const float world_size_x, const float world_size_y
) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) return;
    const float last_movement_x = position_x[idx] - last_position_x[idx];
    const float last_movement_y = position_y[idx] - last_position_y[idx];
    constexpr float velocity_damping = 0.001f;
    float new_position_x = position_x[idx] + last_movement_x + (acceleration_x - last_movement_x * velocity_damping) * (delta_time * delta_time);
    float new_position_y = position_y[idx] + last_movement_y + (acceleration_y - last_movement_y * velocity_damping) * (delta_time * delta_time);

    constexpr float margin = 2.0f;
    if (new_position_x < margin + radius[idx])                     { new_position_x = margin + radius[idx]; }
    else if (new_position_x > world_size_x - margin - radius[idx]) { new_position_x = world_size_x - margin - radius[idx]; }
    if (new_position_y < margin + radius[idx])                     { new_position_y = margin + radius[idx]; }
    else if (new_position_y > world_size_y - margin - radius[idx]) { new_position_y = world_size_y - margin - radius[idx]; }

    last_position_x[idx] = position_x[idx];
    last_position_y[idx] = position_y[idx];
    position_x[idx] = new_position_x;
    position_y[idx] = new_position_y;
}

__global__ void initGridCounts_kernel(int32_t *object_counts, const int grid_count) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < grid_count) {
        object_counts[idx] = 0;
    }
}

__global__ void assignObjectsToGrid_kernel(
    const float *position_x, const float *position_y,
    const int object_count,
    const int world_width,
    const int world_height,
    int32_t *object_index,
    int32_t *object_counts
) {
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= object_count) return;
    const int grid_x = static_cast<int>(floorf(position_x[i]));
    const int grid_y = static_cast<int>(floorf(position_y[i]));
    if (grid_x < 0 || grid_x >= world_width || grid_y < 0 || grid_y >= world_height) return;
    // const int target_idx = grid_y * world_width + grid_x;
    const int target_idx = grid_x * world_height + grid_y;
    const int offset = atomicAdd(&object_counts[target_idx], 1);
    if (offset < num_cell) {
        object_index[target_idx * num_cell + offset] = static_cast<int>(i);
    }
}

__global__ void solveCollisions_kernel(
    float *position_x, float *position_y, const float *radius,
    const int32_t *object_index, const int32_t *object_counts,
    const int grid_count, const int world_width, const int world_height
) {
    const int grid_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (grid_idx >= grid_count) return;
    // const int grid_x = grid_idx % world_width;
    // const int grid_y = grid_idx / world_width;
    const int grid_x = grid_idx / world_height;
    const int grid_y = grid_idx % world_height;
    const int count1 = object_counts[grid_idx];
    for (int dy = -1; dy <= 1; ++dy) {
        for (int dx = -1; dx <= 1; ++dx) {
            const int nx = grid_x + dx;
            const int ny = grid_y + dy;
            if (nx < 0 || nx >= world_width || ny < 0 || ny >= world_height) continue;
            // int nidx = ny * world_width + nx;
            int nidx = nx * world_height + ny;
            int count2 = object_counts[nidx];
            for (int i = 0; i < count1; ++i) {
                int obj1 = object_index[grid_idx * num_cell + i];
                for (int j = 0; j < count2; ++j) {
                    int obj2 = object_index[nidx * num_cell + j];
                    if (obj1 >= 0 && obj2 >= 0 && obj1 != obj2) {
                        float dx = position_x[obj1] - position_x[obj2];
                        float dy = position_y[obj1] - position_y[obj2];
                        float dist2 = dx * dx + dy * dy;
                        float r_sum = radius[obj1] + radius[obj2];
                        if (dist2 < r_sum * r_sum && dist2 > 1e-6f) {
                            float dist = sqrtf(dist2);
                            float delta = 0.5f * (r_sum - dist);
                            float nx = dx / dist;
                            float ny = dy / dist;
                            atomicAdd(&position_x[obj1], nx * delta);
                            atomicAdd(&position_y[obj1], ny * delta);
                            atomicAdd(&position_x[obj2], -nx * delta);
                            atomicAdd(&position_y[obj2], -ny * delta);
                        }
                    }
                }
            }
        }
    }
}

void solveCollisions(const Object *objects, const int32_t *object_index, const int32_t *object_counts, const int grid_count, const int world_width, const int world_height) {
    // int blockSize = 128;
    int gridSize = (grid_count + gpu_block_size - 1) / gpu_block_size;
    solveCollisions_kernel<<<gridSize, gpu_block_size>>>(
        objects->d_position_x, objects->d_position_y, objects->d_radius,
        object_index, object_counts,
        grid_count, world_width, world_height
    );
    // hipDeviceSynchronize();
}

void updateObjects(const Object *objects, const float delta_time, const float world_size_x, const float world_size_y) {
    const int size = objects->size;
    // int blockSize = 256;
    int gridSize = (size + gpu_block_size - 1) / gpu_block_size;
    updateObjects_kernel<<<gridSize, gpu_block_size>>>(
        objects->d_position_x, objects->d_position_y,
        objects->d_last_position_x, objects->d_last_position_y,
        objects->acceleration_x, objects->acceleration_y,
        objects->d_radius,
        size, delta_time, world_size_x, world_size_y
    );
    // hipDeviceSynchronize();
}

void updateGrids(const Object *objects, const int world_width, const int world_height) {
    const int grid_count = grids.grid_count;
    // int blockSize = 256;
    int gridSize = (grid_count + gpu_block_size - 1) / gpu_block_size;
    initGridCounts_kernel<<<gridSize, gpu_block_size>>>(grids.object_counts, grid_count);
    // hipDeviceSynchronize();

    int object_count = objects->size;
    gridSize = (object_count + gpu_block_size - 1) / gpu_block_size;
    assignObjectsToGrid_kernel<<<gridSize, gpu_block_size>>>(
        objects->d_position_x, objects->d_position_y,
        object_count,
        world_width,
        world_height,
        grids.object_index,
        grids.object_counts
    );
    // hipDeviceSynchronize();
}

extern void updatePhysics(Object *objects, const float sub_delta_time, const float sub_steps, const float world_size_x, const float world_size_y) {
    if (objects->size < 0) return;
    objectCopyToDevice(objects);

#ifdef OUTPUT_RESULTS
    hipEvent_t start, end;
    float elapsedTime = 0.0f;

    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start, nullptr);
#endif

    for (int i = 0; i < static_cast<int>(sub_steps); ++i) {
        updateObjects(objects, sub_delta_time, world_size_x, world_size_y);
        updateGrids(objects, static_cast<int>(world_size_x), static_cast<int>(world_size_y));
        solveCollisions(objects, grids.object_index, grids.object_counts, grids.grid_count, static_cast<int>(world_size_x), static_cast<int>(world_size_y));
    }

#ifdef OUTPUT_RESULTS
    hipEventRecord(end, nullptr);
    hipEventSynchronize(end);

    hipEventElapsedTime(&elapsedTime, start, end);
    if (objects->size > particle_min_count) {
        output_file << elapsedTime * 1000 << ",";
    }

    hipEventDestroy(start);
    hipEventDestroy(end);
#endif

    objectCopyToHost(objects);
}

#endif
